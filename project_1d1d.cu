#include "hip/hip_runtime.h"
#include <stdio.h>
#include <sys/time.h>
#include <stdbool.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
//Time complexity is O(2^n)


__global__ void generateSubsetsOnGPU_1D1D_v1(int *MatA, bool *MatB, int nx)
{ 
	unsigned int index = threadIdx.x + blockIdx.x * blockDim.x;
	int sum=0;
    for (int j = 0; j < nx; j++) 
    {
        
        if (index & (1 << j)) 
        {      
            sum += MatA[j];
        }
        
        //sum+= ((index>>j) & 1) *MatA[j];
    }
        //printf("Subset %d: Sum = %d\n", ++count, sum);
        //use one-dim arr to store the sum of subset;
    MatB[index] = sum;
    
}



void have_answer(bool* subsetSums, int target) {
    if (subsetSums[target]) {
        printf("The answer is: Yes\n");
    } else {
        printf("The answer is: No\n");
    }
}


int main(void) {
    
    //testing arr
    int arr[] = {1,2,3};
    int n = sizeof(arr) / sizeof(arr[0]);
    
    int target = 6;

    //set up device
    int dev = 0;
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);
	printf("Using Device %d: %s\n", dev, deviceProp.name);
	hipSetDevice(dev);

    //data-dim nx
    int nx = n;
    int nBytes = (6+1)*sizeof(bool);
    bool  *gpuRef;
    gpuRef = (bool *)malloc(nBytes);

    //start time
    clock_t start, end;
    double cpu_time_used;
    start = clock();

    //initial the user data
    memset(gpuRef, false, nBytes);
    int  *d_arr;
    bool *d_sum_of_subset;
    hipMalloc((void**)&d_arr,n*sizeof(int));
    hipMalloc((void**)&d_sum_of_subset, nBytes);


    //transfer data from host to device
    hipMemcpy(d_arr, arr, n*sizeof(int), hipMemcpyHostToDevice);

    // 1D1D
    int dimx11v1 = 1024; int dimy11v1 = 1;
	dim3 block11v1(dimx11v1, dimy11v1);
	dim3 grid11v1(((1<<n)+block11v1.x-1)/block11v1.x);

    


    generateSubsetsOnGPU_1D1D_v1 <<< grid11v1, block11v1 >>>(d_arr, d_sum_of_subset, nx);
    
    printf("========================\n");

    hipMemcpy(gpuRef, d_sum_of_subset, nBytes, hipMemcpyDeviceToHost);

    
    
    //check the arr number

    //printf("arr[] = %d\n",gpuRef[1048576-1]);//the last number
    //print all the number of gpuRef
    /*
    for(int i=0;i<(1<<n);i++)
    {
        printf("arr[%d] = %d\n", i,gpuRef[i]);
    }
    */
    printf("Find target %d\n",target);
    have_answer(gpuRef, 1<<nx , target);
   

    //stop time
    end = clock();
    cpu_time_used = ((double) (end - start)) / CLOCKS_PER_SEC;
    printf("Time taken: %f seconds\n", cpu_time_used);


    hipFree(d_arr);
    hipFree(d_sum_of_subset);
    free(gpuRef);
    hipDeviceReset();
    return 0;
}