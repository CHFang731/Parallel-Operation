#include "hip/hip_runtime.h"
#include <stdio.h>
#include <sys/time.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

// Time complexity is O(2^n)

__global__ void generateSubsetsOnGPU_1D1D_v1(int *MatA, int *MatB, int nx)
{
    unsigned int index = threadIdx.x + blockIdx.x * blockDim.x;
    int sum = 0;
    for (int j = 0; j < nx; j++)
    {
        if (index & (1 << j))
        {
            sum += MatA[j];
        }
    }
    MatB[index] = sum;
}

void have_answer(int arr[], int size, int target)
{
    for (int i = 0; i < size; i++)
    {
        if (arr[i] == target)
        {
            printf("i = %d\n",i);
            printf("The answer is: Yes\n");
            return;
        }
    }
    printf("The answer is: No\n");
}

int main(void)
{

    // Testing arr
    int arr[] = {1,2,3,4,5,6,7,8,9,10,11,12,13,14,15,16,17,18,19,20,21,22,23,24,25,26,27,28};
    int n = sizeof(arr) / sizeof(arr[0]);
    int target = 2000;

    // Set up device
    int dev = 0;
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);
    printf("Using Device %d: %s\n", dev, deviceProp.name);
    hipSetDevice(dev);

    // Data-dim nx
    int nx = n;
    int nBytes = (1 << nx) * sizeof(int);
    int *gpuRef;

    // Start time
    clock_t start, end;
    double cpu_time_used;
    start = clock();

    // Allocate zero-copy memory 
    hipHostAlloc((void **)&gpuRef, nBytes, hipHostMallocMapped);
    memset(gpuRef, 0, nBytes);


    // Initial the user data
    int *d_sum_of_subset, *d_arr;
    hipMalloc((void **)&d_arr, n * sizeof(int));
    hipMalloc((void **)&d_sum_of_subset, nBytes);

    // Transfer data from host to device
    hipMemcpy(d_arr, arr, n * sizeof(int), hipMemcpyHostToDevice);
    hipHostGetDevicePointer((void**)&d_sum_of_subset, (void *)gpuRef, 0);

    // 1D1D
    int dimx11v1 = 1024;
    int dimy11v1 = 1;
    dim3 block11v1(dimx11v1, dimy11v1);
    dim3 grid11v1(((1 << n) + block11v1.x - 1) / block11v1.x);

    generateSubsetsOnGPU_1D1D_v1<<<grid11v1, block11v1>>>(d_arr, d_sum_of_subset, nx);
    printf("Find target %d\n",target);
    printf("========================\n");

    //hipMemcpy(gpuRef, d_sum_of_subset, nBytes, hipMemcpyDeviceToHost);

    // Check the arr number
    have_answer(gpuRef, 1 << nx, target);

    // Stop time
    end = clock();
    cpu_time_used = ((double)(end - start)) / CLOCKS_PER_SEC;
    printf("Time taken: %f seconds\n", cpu_time_used);

    // Free memory
    hipHostFree(gpuRef);
    hipFree(d_arr);
    hipFree(d_sum_of_subset);
    hipDeviceReset();
    return 0;
}
