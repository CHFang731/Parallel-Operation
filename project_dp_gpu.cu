#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>

#define BLOCK_SIZE 256

__global__ void subsetSum(int *d_arr, bool *d_subsetSums, int n, int target) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < (1 << n)) {  // Iterate over all possible subsets using bitmasking
        int subsetSum = 0;
        for (int i = 0; i < n; ++i) {
            if (idx & (1 << i)) {  // Check if ith element is in the subset
                subsetSum += d_arr[i];
            }
        }
        if (subsetSum <= target) {
            d_subsetSums[subsetSum] = true;
        }
    }
}

// Function to get current time in microseconds
long long getCurrentTime() {
    struct timeval tv;
    gettimeofday(&tv, NULL);
    return tv.tv_sec * 1000000LL + tv.tv_usec;
}

int main(int argc, char *argv[]) {
    int arr[] = {1, 2, 3, 4,5,6,7,8,9,10,11,12,13,14,15,16,17,18,19,20,21,22,23,24,25,26,27,28};
    int n = sizeof(arr) / sizeof(arr[0]);
    int target = 2000;

    // Allocate memory on the device
    int *d_arr;
    bool *d_subsetSums;
    hipMalloc((void **)&d_arr, n * sizeof(int));
    hipMalloc((void **)&d_subsetSums, (target + 1) * sizeof(bool));

    // Copy input data from host to device
    hipMemcpy(d_arr, arr, n * sizeof(int), hipMemcpyHostToDevice);

    // Start timing
    long long startTime = getCurrentTime();

    // Launch kernel
    int numBlocks = (1 << n) / BLOCK_SIZE + 1;
    subsetSum<<<numBlocks, BLOCK_SIZE>>>(d_arr, d_subsetSums, n, target);

    // Stop timing
    hipDeviceSynchronize();
    long long endTime = getCurrentTime();
    double totalTime = (endTime - startTime) / 1000000.0;  // Convert microseconds to seconds

    // Copy result back to host
    bool *subsetSums = (bool *)malloc((target + 1) * sizeof(bool));
    hipMemcpy(subsetSums, d_subsetSums, (target + 1) * sizeof(bool), hipMemcpyDeviceToHost);

    // Check if there exists a subset with sum equal to target
    if (subsetSums[target]) {
        printf("The answer is: Yes\n");
    } else {
        printf("The answer is: No\n");
    }

    printf("Time taken: %f seconds\n", totalTime);

    // Cleanup
    free(subsetSums);
    hipFree(d_arr);
    hipFree(d_subsetSums);

    return 0;
}
